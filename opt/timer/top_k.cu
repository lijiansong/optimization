#include "hip/hip_runtime.h"
/*
Members:
201618013229011	李坚松
201618013229015	刘刚
201618013229006	黄若然
201618013229012	李琨
201618013229014	刘伯然
*/
#include <vector>
#include <queue>
#include <limits.h>
#include <float.h>
#include <functional>
#include <fstream>

#include <assert.h>
#include <sys/time.h>
#include <unistd.h>
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <hip/hip_runtime.h>
//#include "cuPrintf.cu"

using namespace std;

typedef vector<float> vf;
typedef priority_queue<float,vector<float>,greater<float> > pq;

#define MAX_THREAD_PER_BLOCK 1024

static void HandleError(hipError_t err, const char *file, int line )
{
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n",hipGetErrorString(err), file, line );
		exit(EXIT_FAILURE);
	}
}

#define HANDLE_ERROR( err ) (HandleError(err, __FILE__, __LINE__))
#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}
		
void get_device_info()
{
	hipDeviceProp_t prop;
	int count;
	HANDLE_ERROR(hipGetDeviceCount(&count));
	for (int i = 0; i < count; ++i)
	{
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
		printf("  --- Info of device %d ---\n",i );
		printf("device name : %s.\n", prop.name);
		printf("total global mem : %d.\n", prop.totalGlobalMem);
		printf("shared mem per block : %d.\n", prop.sharedMemPerBlock);
		printf("registers per block : %d.\n", prop.regsPerBlock);
		printf("threrads in warp : %d.\n", prop.warpSize);
		printf("mem pitch : %d.\n", prop.memPitch);
		printf("max threads per block : %d.\n", prop.maxThreadsPerBlock);
		printf("max threads dimensions : (%d %d %d).\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("max grid dimensions : (%d %d %d).\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("total const Mem : %d.\n", prop.totalConstMem);
		printf("major.minor : %d.%d.\n", prop.major, prop.minor);
		printf("clock rate : %d.\n", prop.clockRate);
		printf("texture alignment : %d.\n", prop.textureAlignment);
		printf("device overlap : %d.\n", prop.deviceOverlap);
		printf("multiprocessor count : %d.\n", prop.multiProcessorCount);
		printf("\n");
	}
}

int partition(vf &data,int start, int end)
{
	int i=start,j=end;
	float tmp;
	if(start<end)
	{
		tmp=data[start];
		while(i!=j)
		{
			while(j>i&&data[j]-tmp<=1e-9)	--j;
			data[i]=data[j];
			while(j>i&&data[i]-tmp>1e-9)	++i;
			data[j]=data[i];
			
		}
		data[i]=tmp;
	}
	return i;
}

void _top_k(vf &data,int k)
{
	int start=0,end=data.size()-1;
	int index=partition(data,start,end);
	while(index!=k-1)
	{
		if(index > k-1)
		{
			end=index-1;
			index=partition(data,start,end);
		}
		else
		{
			start=index+1;
			index=partition(data,start,end);
		}
	}
}

void heap_init(vf &data, pq &heap, int k)
{
	if(k > data.size())	return ;
	for(int i=0;i<k;++i)
	{
		heap.push(data[i]);
	}
}

void top_k(vf &data, pq &heap, int k)
{
	int len=data.size();
	if(k > len)	return ;
	for(int i=k;i<len;++i)
	{
		if(heap.top()-data[i]<1e-9)
		{
			heap.pop();
			heap.push(data[i]);
		}
	}
}

__device__ void selection_sort(float *dev_data, int left, int right)
{
	for (int i = left; i <= right; ++i)
	{
		float max_val=dev_data[i];
		int max_idx=i;
		for (int j = i+1; j <= right; ++j)
		{
			float val_j=dev_data[j];
			if(val_j - max_val > 1e-9)
			{
				max_idx=j;
				max_val=val_j;
			}
		}
		if (i != max_idx)
		{
			dev_data[max_idx]=dev_data[i];
			dev_data[i]=max_val;
		}
	}
}

__global__ void topk_kernel(float *dev_data, float *dev_result,const int &k,const int &data_len, const int &width)
{
	int tx=threadIdx.x;
	//cuPrintf("-----------tx is:%d\n", tx);
	//printf("-----------tx is:%d\n", tx);
	//printf("-------------------%d",tx*width+width);
	if (tx*width+width <= data_len)
	{
		selection_sort(dev_data,tx*width,tx*width+width-1);
	}
	else
	{
		selection_sort(dev_data,tx*width,data_len-1);
	}
	
	__shared__ float heap[MAX_THREAD_PER_BLOCK];
	__shared__ int res_index;
	__shared__ int local_index[MAX_THREAD_PER_BLOCK];
	assert(tx!=0);
	if (tx==0)
	{
		res_index=0;
		for (int i = 0; i < MAX_THREAD_PER_BLOCK; ++i)
		{
			heap[i]=FLT_MIN;
			local_index[i]=0;
		}
	}
	__syncthreads();
	
	for (int i = 0; i < k; ++i)
	{
		if (local_index[tx] < width)
		{
			//printf("-------------%f",dev_data[tx*width+local_index[tx]]);
			heap[tx]=dev_data[tx*width+local_index[tx]];
		}
		else
		{
			heap[tx]=FLT_MIN;
		}
		__syncthreads();

		if(tx==0)
		{
			float max_val=heap[0];
			int max_index=0;
			for (int i = 1; i < MAX_THREAD_PER_BLOCK; ++i)
			{
				if (heap[i]-max_val > 1e-9)
				{
					max_val=heap[i];
					max_index=i;
				}
			}
			++local_index[max_index];
			//cuPrintf("-------------------------max_val is:%f\n", max_val);
			dev_result[res_index]=max_val;
			++res_index;
		}
		__syncthreads();
	}
}

void topk_device(const vf &data,const int &k,float *result)
{
	float *dev_data;
	size_t len=data.size();
	//printf("len--------------%d\n", len);
	HANDLE_ERROR(hipMalloc((void**)&dev_data,len*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(dev_data,&data[0],len*sizeof(float),hipMemcpyHostToDevice));

	float *dev_result;
	HANDLE_ERROR(hipMalloc((void**)&dev_result,k*sizeof(float)));

	int width=len/MAX_THREAD_PER_BLOCK;
	//printf("-----------width %d\n", width);

	//cudaPrintfInit(); 

	dim3 dimBlock(MAX_THREAD_PER_BLOCK,1);
	dim3 dimGrid(1,1);
	topk_kernel<<<dimGrid,dimBlock>>>(dev_data,dev_result,k,len,width);

	//cudaPrintfDisplay(stdout, true); 
	//cudaPrintfEnd(); 

	hipMemcpy(result,dev_result,k*sizeof(float),hipMemcpyDeviceToHost);

	hipFree(dev_data);
	hipFree(dev_result);
}

void parse_data(const char* filename,vf &data)
{
	ifstream infile;
    string line;
    infile.open(filename);
    if(infile.is_open())
    {
    	while(getline(infile,line,'\n'))
    	{
    		data.push_back((float)atof(line.c_str()));
    	}
    }
    infile.close();
}

int main(int argc, char const *argv[])
{
	if(argc!=3)	return -1;
	
	//get_device_info();
	struct timeval _start,_end;
	double diff;
	
	vf data;
	pq heap;
	parse_data(argv[1],data);
	int k=atoi(argv[2]);

	float *result=(float*)malloc(k*sizeof(float));

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	//cudaPrintfInit(); 
	
	topk_device(data,k,result);
	
	//cudaPrintfDisplay(stdout, true); 
	//cudaPrintfEnd(); 
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,start,stop);
	
	gettimeofday(&_start,NULL);
	//heap_init(data,heap,k);
    //top_k(data,heap,k);
	_top_k(data,k);
	gettimeofday(&_end,NULL);
    diff = (1000000 * (_end.tv_sec-_start.tv_sec)+ _end.tv_usec-_start.tv_usec)/10000.0;
	
	printf("----------------top %d----------------\n",k);
	//for (int i = 0; i < k; ++i)
	//{
	//	printf("%f ", result[i]);
	//}
	//printf("\n");
	//printf("\n");
	//for(int i=0;i<k;++i)
	//{
        //printf("%f ",heap.top());
        //heap.pop();
   // }
   for(int i=0;i<k;++i)	printf("%f ",data[i]);
	printf("\n");
	printf("time consuming: %f ms\n", diff);
	//printf("time consuming: %f ms\n", elapsedTime);

	free(result);
	return 0;
}
